#include "hip/hip_runtime.h"
/****************************************************************************
 *
 * cuda-stencil1d.cu - 1D stencil example with CUDA
 *
 * Based on the examples from the CUDA toolkit documentation
 * http://docs.nvidia.com/cuda/cuda-c-programming-guide/
 *
 * Last updated in 2017 by Moreno Marzolla <moreno.marzolla(at)unibo.it>
 *
 * ---------------------------------------------------------------------------
 *
 * This implementation does not use shared memory.
 *
 * Compile with:
 * nvcc cuda-stencil1d.cu -o cuda-stencil1d
 *
 * Run with:
 * ./cuda-stencil1d
 *
 ****************************************************************************/

#include <stdio.h>
#include <stdlib.h>
#include <assert.h>

#define BLKDIM 1024
#define RADIUS 3
#define N (2048*2048)

double gettime( void )
{
    struct timespec ts;
    clock_gettime(CLOCK_MONOTONIC, &ts );
    return (ts.tv_sec + (double)ts.tv_nsec / 1e9);
}

__global__ void stencil1dNoShared(int *in, int *out) 
{
    const int index = threadIdx.x + blockIdx.x * blockDim.x + RADIUS;
    int result = 0, offset;
    for (offset = -RADIUS ; offset <= RADIUS ; offset++) {
        result += in[index + offset];
    }
    /* Store the result */
    out[index] = result;
}

__global__ void stencil1dWithShared(int *in, int *out) 
{
    __shared__ int temp[BLKDIM + 2 * RADIUS];
    const int gindex = threadIdx.x + blockIdx.x * blockDim.x + RADIUS;
    const int lindex = threadIdx.x + RADIUS;
    int result = 0, offset;
    /* Read input elements into shared memory */
    temp[lindex] = in[gindex];
    if (threadIdx.x < RADIUS) {
        temp[lindex - RADIUS] = in[gindex - RADIUS];
        temp[lindex + blockDim.x] = in[gindex + blockDim.x];
    }
    __syncthreads(); 
    /* Apply the stencil */
    for (offset = -RADIUS ; offset <= RADIUS ; offset++) {
        result += temp[lindex + offset];
    }
    /* Store the result */
    out[gindex] = result;
}

int main( void ) 
{
    int *h_in, *h_out;	  /* host copies of in and out */
    int *d_in, *d_out;	  /* device copies of in and out */
    int i;
    const size_t size = (N+2*RADIUS)*sizeof(int);
    double tstart,tend;

    assert( N % BLKDIM == 0 );

    /* Allocate space for device copies of d_in and d_out */
    hipMalloc((void **)&d_in, size);
    hipMalloc((void **)&d_out, size);
    /* Allocate space for host copies of h_in and h_out */
    h_in = (int*)malloc(size);
    h_out = (int*)malloc(size);
    /* Set all elements of h_in to one */
    for (i=0; i<N+2*RADIUS; i++) {
        h_in[i] = 1;
    }
    /* Copy input to device */
    hipMemcpy(d_in, h_in, size, hipMemcpyHostToDevice);

    /* Launch stencil1dNoShared() kernel on GPU */
    tstart=gettime();
    stencil1dNoShared<<<(N + BLKDIM-1)/BLKDIM, BLKDIM>>>(d_in, d_out);
    hipDeviceSynchronize();
    tend=gettime();
    printf("Elapsed time in seconds for 1D stencil without shared memory: %f\n", tend-tstart);

    /* Copy result back to host */
    hipMemcpy(h_out, d_out, size, hipMemcpyDeviceToHost);
    /* Check the result */
    for (i=RADIUS; i<N+RADIUS; i++) {
        if ( h_out[i] != 7 ) {
            fprintf(stderr, "Error at index %d: h_out[%d] == %d, expected 7\n", i, i, h_out[i]);
            return EXIT_FAILURE;
        }
    }
    printf("Test OK\n");
    
    /* Launch stencil1dWithShared() kernel on GPU */
    tstart=gettime();
    stencil1dWithShared<<<(N + BLKDIM-1)/BLKDIM, BLKDIM>>>(d_in, d_out);
    hipDeviceSynchronize();
    tend=gettime();
    printf("Elapsed time in seconds for 1D stencil with shared memory: %f\n", tend-tstart);

    /* Copy result back to host */
    hipMemcpy(h_out, d_out, size, hipMemcpyDeviceToHost);
    /* Check the result */
    for (i=RADIUS; i<N+RADIUS; i++) {
        if ( h_out[i] != 7 ) {
            fprintf(stderr, "Error at index %d: h_out[%d] == %d, expected 7\n", i, i, h_out[i]);
            return EXIT_FAILURE;
        }
    }
    printf("Test OK\n");
    /* Cleanup */
    free(h_in); free(h_out);
    hipFree(d_in); hipFree(d_out);
    return EXIT_SUCCESS;
}
