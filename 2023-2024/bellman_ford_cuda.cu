#include "hip/hip_runtime.h"
#include "bellman_ford.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <limits.h>

#define INF INT_MAX

__global__ void relaxEdges(Edge *edgeList, int *dist, int edges) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < edges) {
        int u = edgeList[idx].src;
        int v = edgeList[idx].dest;
        int weight = edgeList[idx].weight;
        if (dist[u] != INF && dist[u] + weight < dist[v]) {
            dist[v] = dist[u] + weight;
        }
    }
}

void bellman_ford_cuda(int vertices, int edges, Edge *edgeList, int source) {
    int *dist = (int *)malloc(vertices * sizeof(int));
    for (int i = 0; i < vertices; i++) {
        dist[i] = INF;
    }
    dist[source] = 0;

    Edge *d_edgeList;
    int *d_dist;

    hipMalloc(&d_edgeList, edges * sizeof(Edge));
    hipMalloc(&d_dist, vertices * sizeof(int));

    hipMemcpy(d_edgeList, edgeList, edges * sizeof(Edge), hipMemcpyHostToDevice);
    hipMemcpy(d_dist, dist, vertices * sizeof(int), hipMemcpyHostToDevice);

    int blockSize = 256;
    int numBlocks = (edges + blockSize - 1) / blockSize;

    for (int i = 1; i <= vertices - 1; i++) {
        relaxEdges<<<numBlocks, blockSize>>>(d_edgeList, d_dist, edges);
        hipDeviceSynchronize();
    }

    hipMemcpy(dist, d_dist, vertices * sizeof(int), hipMemcpyDeviceToHost);

    for (int i = 0; i < vertices; i++) {
        printf("Vertex %d: %d\n", i, dist[i]);
    }

    hipFree(d_edgeList);
    hipFree(d_dist);
    free(dist);
}
